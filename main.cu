#include "hip/hip_runtime.h"
//==============================================================================================
// Originally written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is
// distributed without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication
// along with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==============================================================================================

#include "rtweekend.h"

#include "camera.h"
#include "color.h"
#include "hittable_list.h"
#include "material.h"
#include "sphere.h"
#include <chrono>

#include <iostream>

__device__ int foo(const ray& r, const hittable_list* world, int d) {
    hit_record* hr = new hit_record;
    if (d == 0) {
        return 0;
    }
    return foo(r, world, d - 1) + 1;
}

__device__ color ray_color(ray r, const hittable_list* world, int depth) {
    hit_record rec;
    color accu(1, 1, 1);  // accumulation of attenuation
    for (int i = depth; i > 0; i--) {
        if (world->hit(r, 0.001, infinity, &rec)) {
            ray scattered;
            color attenuation;
            if (rec.mat_ptr->scatter(r, rec, &attenuation, &scattered)) {
                accu = accu * attenuation;
                r = scattered;
                continue;
            }
            return color(0, 0, 0);
        } else {
            vec3 unit_direction = unit_vector(r.direction());
            auto t = 0.5 * (unit_direction.y() + 1.0);
            return accu * ((1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0));
        }
    }

    return color(0, 0, 0);

    // hit_record rec;
    // // If we've exceeded the ray bounce limit, no more light is gathered.
    // if (depth <= 0) {
    //     return color(0, 0, 0);
    // }
    // if (world->hit(r, 0.001, infinity, &rec)) {
    //     ray scattered;
    //     color attenuation;
    //     if (rec.mat_ptr->scatter(r, rec, &attenuation, &scattered)) {
    //         if (depth == 47) {
    //             return color(0, 0, 0);
    //         }
    //         return attenuation * ray_color(scattered, world, depth - 1);
    //     }
    //     return color(0, 0, 0);
    // }

    // vec3 unit_direction = unit_vector(r.direction());
    // auto t = 0.5 * (unit_direction.y() + 1.0);
    // return (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

__global__ void random_scene(hittable_list* world) {
    world->objects = new sphere*[500];
    world->tail = 0;

    random_init();

    auto ground_material = new material(1);
    ground_material->setup1(color(0.5, 0.5, 0.5));
    world->add(new sphere(point3(0, -1000, 0), 1000, ground_material));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            auto choose_mat = random_double();
            point3 center(a + 0.9 * random_double(), 0.2, b + 0.9 * random_double());

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                material* sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color::random() * color::random();
                    sphere_material = new material(1);
                    sphere_material->setup1(albedo);
                    world->add(new sphere(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color::random(0.5, 1);
                    auto fuzz = random_double(0, 0.5);
                    sphere_material = new material(2);
                    sphere_material->setup2(albedo, fuzz);
                    world->add(new sphere(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = new material(3);
                    sphere_material->setup3(1.5);
                    world->add(new sphere(center, 0.2, sphere_material));
                }
            }
        }
    }
    auto material1 = new material(3);
    material1->setup3(1.5);
    world->add(new sphere(point3(0, 1, 0), 1.0, material1));

    auto material2 = new material(1);
    material2->setup1(color(0.4, 0.2, 0.1));
    world->add(new sphere(point3(-4, 1, 0), 1.0, material2));

    auto material3 = new material(2);
    material3->setup2(color(0.7, 0.6, 0.5), 0.0);
    world->add(new sphere(point3(4, 1, 0), 1.0, material3));
}

__global__ void ray_trace_pixel(
    camera cam, hittable_list* world, unsigned char* out_image) {

    const int image_width = 1024;
    const int image_height = 576;
    const int samples_per_pixel = 10;
    const int max_depth = 50;

    for (int k = 0; k < 4; k++) {
        int i = threadIdx.x * 4 + k, j = blockIdx.x;
        color pixel_color(0, 0, 0);
        for (int s = 0; s < samples_per_pixel; ++s) {
            auto u = (i + random_double()) / (image_width - 1);
            auto v = (j + random_double()) / (image_height - 1);
            ray r = cam.get_ray(u, v);
            // printf("%d\n", foo(r, world, 50));
            pixel_color += ray_color(r, world, max_depth);
        }

        pixel_color.postprocessing(samples_per_pixel);
        out_image[3 * (image_width * (image_height - 1 - j) + i) + 0] = pixel_color.f[2];
        out_image[3 * (image_width * (image_height - 1 - j) + i) + 1] = pixel_color.f[1];
        out_image[3 * (image_width * (image_height - 1 - j) + i) + 2] = pixel_color.f[0];
    }
}

int main(int argc, char** argv) {

    // Image
    const auto aspect_ratio = 16.0 / 9.0;
    const int image_width = 1024;
    const int image_height = 576;
    // const int samples_per_pixel = 10;
    // const int max_depth = 50;

    unsigned char* out_image = (unsigned char*)malloc(image_height * image_width * 3 * sizeof(unsigned char));
    unsigned char* dev_out_image;
    hipMalloc(&dev_out_image, image_height * image_width * 3 * sizeof(unsigned char));

    hittable_list* world;
    hipMalloc(&world, sizeof(hittable_list));
    random_scene<<<1, 1>>>(world);
    hipDeviceSynchronize();

    // Camera

    point3 lookfrom(13, 2, 3);
    point3 lookat(0, 0, 0);
    vec3 vup(0, 1, 0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;

    camera cam(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus);

    // Render
    // std::chrono::duration<double> t;
    // auto startTime = std::chrono::steady_clock::now(), endTime = startTime;

    ray_trace_pixel<<<image_height, 256>>>(cam, world, dev_out_image);
    hipDeviceSynchronize();

    // endTime = std::chrono::steady_clock::now();
    // t = endTime - startTime;
    // std::cout << t.count() << "secs." << std::endl;

    hipMemcpy(out_image, dev_out_image, image_height * image_width * 3 * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    write_png(argv[1], out_image, image_height, image_width, 3);
    std::cerr << "\nDone.\n";
}
